#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void addVectors(float* a, float* b, float* c, int N)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < N )
    {
        c[index] = a[index] + b[index];
    }
}

int main()
{
    hipError_t error = hipSuccess;
    int N = 50000;
    size_t sizeN = N * sizeof(float);
    
    float* h_a = (float*)malloc(sizeN);
    float* h_b = (float*)malloc(sizeN);
    float* h_c = (float*)malloc(sizeN);

    for(int i=0; i<N; i++)
    {
        h_a[i] = rand() / (float)RAND_MAX;
        h_b[i] = rand() / (float)RAND_MAX;
    }

    float* d_a=NULL, *d_b=NULL, *d_c=NULL;

    error = hipMalloc(&d_a, sizeN); 
    if(error != hipSuccess)
    {
        std::cerr << "failed to allocate gpu memory to device A :" << hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }

    error = hipMalloc(&d_b, sizeN); 
    if(error != hipSuccess)
    {
        std::cerr << "failed to allocate gpu memory to device B :" << hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }

    error = hipMalloc(&d_c, sizeN); 
    if(error != hipSuccess)
    {
        std::cerr << "failed to allocate gpu memory to device C :" << hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_a, h_a, sizeN, hipMemcpyHostToDevice);
    if(error != hipSuccess)
    {
        std::cerr << " failed to copy memory from host_A to device_A : " hipGetErrorString(error) <<std::endl;
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_b, h_b, sizeN, hipMemcpyHostToDevice);
    if(error != hipSuccess)
    {
        std::cerr << " failed to copy memory from host_B to device_B : " hipGetErrorString(error) <<std::endl;
        exit(EXIT_FAILURE);
    }

    int blockDimValue = 256;
    int gridDimValue = (N + blockDimValue - 1)/blockDimValue;

    addVectors <<< gridDimValue, blockDimValue >>> (d_a, d_b, d_c, N);


    error = hipMemcpy(h_c, d_c, sizeN, hipMemcpyDeviceToHost);
    if(error != hipSuccess)
    {
        std::cerr << " failed to copy memory from device_C to host_C : " hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }

    hipFree(d_a);
    if(error != hipSuccess)
    {
        std::cerr << "Failed to free device memory of device A : " hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }

    hipFree(d_b);
    if(error != hipSuccess)
    {
        std::cerr << "Failed to free device memory of device B : " hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }

    hipFree(d_c);
    if(error != hipSuccess)
    {
        std::cerr << "Failed to free device memory of device C : " hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
